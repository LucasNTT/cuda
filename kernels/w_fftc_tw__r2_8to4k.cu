#include "hip/hip_runtime.h"
/*
Copyright (C) 2025 Guillaume P. Hérault (https://github.com/LucasNTT/LucasNTT)

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is furnished to do so, subject to the following condition:

The above copyright notice and this permission notice shall be included in all copies or substantial portions of the Software.
*/

#include "hip/hip_runtime.h"
#include ""
#include "../data_contexts.cuh"
#include "../kernel.cuh"
#include "../common/common.cuh"
#include "../common/arithmetics.cuh"
#include "../core/fft_r2.cuh"
#include "w_fftc_tw__r2_8to4k.cuh"

template<uint32_t log_n, uint32_t bfliesPerThread> __global__ void weight_fftcol_twiddle__radix2_8to4k(uint64_t* xinput, uint32_t stride, const uint64_t* __restrict__ roots, const uint64_t* __restrict__ weights, const uint64_t* __restrict__ twiddleFactors) {
	extern __shared__ uint64_t s[];
	uint64_t* s_roots = (uint64_t*)(&s[1 << log_n]);

	// 1. Transfer from global memory to shared memory
	int tid = threadIdx.x * stride + blockIdx.x; // non-coalesced access		
	constexpr uint32_t gap = (1 << log_n) / bfliesPerThread / 2;
#pragma unroll	
	for (int i = 0; i < 2 * bfliesPerThread; i++) {
		s[threadIdx.x + i * gap] = mul32_Mod(xinput[tid + i * gap * stride], weights[tid + i * gap * stride]);
	}
#pragma unroll	
	for (int i = 0; i < bfliesPerThread; i++) {
		s_roots[threadIdx.x + i * gap] = roots[threadIdx.x + i * gap];
	}
	__syncthreads();

	// 2. Compute the smalls FFT in shared mem
	fft_radix2_pease_inplace<log_n, bfliesPerThread, false>(s, s_roots);
	__syncthreads();

	// 3. Transfer from shared memory to global memory
#pragma unroll	
	for (int i = 0; i < 2 * bfliesPerThread; i++) {
		xinput[tid + i * gap * stride] = mul_Mod(s[threadIdx.x + i * gap], twiddleFactors[tid + i * gap * stride]);
	}
}


w_fftc_tw__r2_8to4k::w_fftc_tw__r2_8to4k(LucasPRPData* lucasPRPData, Length* length, uint32_t count, uint32_t gap)
	: NttFactor(lucasPRPData, length, count, gap) {
	Name = "w_fftc_tw__r2_8to4k";
}

bool w_fftc_tw__r2_8to4k::Initialize() {
	if (data->length->log3_n != 0 || data->length->log5_n != 0 || data->length->log2_n < 3 || data->length->log2_n > 12)
		return false;
	return NttFactor::Initialize();
}

void w_fftc_tw__r2_8to4k::Run() {
	switch (100 * data->length->log2_n + bfliesPerThread) {
	case 301: weight_fftcol_twiddle__radix2_8to4k<3, 1> <<< data->count, data->length->n / 2, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 302: weight_fftcol_twiddle__radix2_8to4k<3, 2> <<< data->count, data->length->n / 4, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 304: weight_fftcol_twiddle__radix2_8to4k<3, 4> <<< data->count, data->length->n / 8, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 308: weight_fftcol_twiddle__radix2_8to4k<3, 8> <<< data->count, data->length->n / 16, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 401: weight_fftcol_twiddle__radix2_8to4k<4, 1> <<< data->count, data->length->n / 2, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 402: weight_fftcol_twiddle__radix2_8to4k<4, 2> <<< data->count, data->length->n / 4, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 404: weight_fftcol_twiddle__radix2_8to4k<4, 4> <<< data->count, data->length->n / 8, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 408: weight_fftcol_twiddle__radix2_8to4k<4, 8> <<< data->count, data->length->n / 16, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 501: weight_fftcol_twiddle__radix2_8to4k<5, 1> <<< data->count, data->length->n / 2, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 502: weight_fftcol_twiddle__radix2_8to4k<5, 2> <<< data->count, data->length->n / 4, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 504: weight_fftcol_twiddle__radix2_8to4k<5, 4> <<< data->count, data->length->n / 8, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 508: weight_fftcol_twiddle__radix2_8to4k<5, 8> <<< data->count, data->length->n / 16, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 601: weight_fftcol_twiddle__radix2_8to4k<6, 1> <<< data->count, data->length->n / 2, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 602: weight_fftcol_twiddle__radix2_8to4k<6, 2> <<< data->count, data->length->n / 4, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 604: weight_fftcol_twiddle__radix2_8to4k<6, 4> <<< data->count, data->length->n / 8, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 608: weight_fftcol_twiddle__radix2_8to4k<6, 8> <<< data->count, data->length->n / 16, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 701: weight_fftcol_twiddle__radix2_8to4k<7, 1> <<< data->count, data->length->n / 2, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 702: weight_fftcol_twiddle__radix2_8to4k<7, 2> <<< data->count, data->length->n / 4, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 704: weight_fftcol_twiddle__radix2_8to4k<7, 4> <<< data->count, data->length->n / 8, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 708: weight_fftcol_twiddle__radix2_8to4k<7, 8> <<< data->count, data->length->n / 16, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 801: weight_fftcol_twiddle__radix2_8to4k<8, 1> <<< data->count, data->length->n / 2, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 802: weight_fftcol_twiddle__radix2_8to4k<8, 2> <<< data->count, data->length->n / 4, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 804: weight_fftcol_twiddle__radix2_8to4k<8, 4> <<< data->count, data->length->n / 8, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 808: weight_fftcol_twiddle__radix2_8to4k<8, 8> <<< data->count, data->length->n / 16, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 901: weight_fftcol_twiddle__radix2_8to4k<9, 1> <<< data->count, data->length->n / 2, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 902: weight_fftcol_twiddle__radix2_8to4k<9, 2> <<< data->count, data->length->n / 4, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 904: weight_fftcol_twiddle__radix2_8to4k<9, 4> <<< data->count, data->length->n / 8, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 908: weight_fftcol_twiddle__radix2_8to4k<9, 8> <<< data->count, data->length->n / 16, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 1001: weight_fftcol_twiddle__radix2_8to4k<10, 1> <<< data->count, data->length->n / 2, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 1002: weight_fftcol_twiddle__radix2_8to4k<10, 2> <<< data->count, data->length->n / 4, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 1004: weight_fftcol_twiddle__radix2_8to4k<10, 4> <<< data->count, data->length->n / 8, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 1008: weight_fftcol_twiddle__radix2_8to4k<10, 8> <<< data->count, data->length->n / 16, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 1101: weight_fftcol_twiddle__radix2_8to4k<11, 1> <<< data->count, data->length->n / 2, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 1102: weight_fftcol_twiddle__radix2_8to4k<11, 2> <<< data->count, data->length->n / 4, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 1104: weight_fftcol_twiddle__radix2_8to4k<11, 4> <<< data->count, data->length->n / 8, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 1108: weight_fftcol_twiddle__radix2_8to4k<11, 8> <<< data->count, data->length->n / 16, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 1202: weight_fftcol_twiddle__radix2_8to4k<12, 2> <<< data->count, data->length->n / 4, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 1204: weight_fftcol_twiddle__radix2_8to4k<12, 4> <<< data->count, data->length->n / 8, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	case 1208: weight_fftcol_twiddle__radix2_8to4k<12, 8> <<< data->count, data->length->n / 16, 3 * data->length->n / 2 * sizeof(uint64_t), common::stream >>> (lucasPRPData->x, data->stride, h_roots[0], lucasPRPData->weights, twiddleFactors); break;
	}
	cudaCheckErrors("Kernel launch failed: weight_fftcol_twiddle__radix2_8to4k");
}
