#include "hip/hip_runtime.h"
/*
Copyright (C) 2025 Guillaume P. Hérault (https://github.com/LucasNTT/LucasNTT)

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is furnished to do so, subject to the following condition:

The above copyright notice and this permission notice shall be included in all copies or substantial portions of the Software.
*/

#include "hip/hip_runtime.h"
#include ""
#include "../data_contexts.cuh"
#include "../kernel.cuh"
#include "../common/common.cuh"
#include "../common/arithmetics.cuh"
#include "../common/memory_tracker.cuh"
#include "../core/atomic.cuh"
#include "../core/transpose_oop.cuh"
#include "r2b__tr.cuh"

///
/// Reduce to variable base 
/// with carry propagation
/// in-place, using global memory, one thread per element 
///
__global__ void reduceToBase_transposed(uint64_t* x, const uint8_t* __restrict__ widths, uint32_t stride)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int maxId = gridDim.x* blockDim.x;

	uint8_t w0 = widths[tid];
	uint64_t mask = (((uint64_t)1) << w0) - 1;
	uint64_t a = 0;
	uint8_t w = 0;

	// Keep the w least significant bits and send the most significant bits to the neighbours
	// First, we need to do x[tid] = (x[tid] + a) & mask
	// But because of the carry propagation, each x[tid] may be overwritten by several threads
	// So use an atomic function to make sure we write the correct data
	uint64_t old = atomicAddAnd(&x[tid], a, mask);

	// Then do carry propagation of the addition
	uint64_t carry = (old + a) >> w0;
	while (carry) {
		tid += stride;
		if (tid >= maxId) {
			uint32_t col = tid & (stride - 1);
			tid = (col + 1 >= stride) ? 0 : col + 1;
		}
			
		w = widths[tid];
		mask = (((uint64_t)1) << w) - 1;

		old = atomicAddAnd(&x[tid], carry, mask); // meaning old = x[tid]; x[tid] = (x[tid] + carry) & mask;
		carry = (old + carry) >> w;
	}
}

r2b__tr::r2b__tr(LucasPRPData* lucasPRPData, Length* length, uint32_t count, uint32_t stride)
	: Kernel(lucasPRPData, length, count, stride) {
	Name = "r2b__tr";
}

bool r2b__tr::Initialize() {
	try {
		cudaMallocTracked((void**)&widths, lucasPRPData->totalLength->n * sizeof(uint64_t));
		cudaCheckErrors("hipMalloc (local widths) failed!");
	}
	catch (...) {
		return false;
	}
	dim3 threads_tr(16, 16);
	dim3 blocks_tr(data->count / 16, data->stride / 16);
	transpose_outofplace<16, 1, TranspositionOperation::NO_MUL, uint8_t> <<< blocks_tr, threads_tr, 0, common::stream >>> (lucasPRPData->widths, widths, nullptr);
	cudaCheckErrors("hipFree (local widths transpose) failed!");
	return Kernel::Initialize();
}

void r2b__tr::Finalize() {
	cudaFreeTracked(widths);
	cudaCheckErrors("hipFree (local widths) failed!");
	Kernel::Finalize();
}

void r2b__tr::Run() {
	/*  Reduce to irrational base with carry propagation */
	reduceToBase_transposed <<< blocks, threads, 0, common::stream >>> (lucasPRPData->x, widths, data->stride);
	cudaCheckErrors("Kernel launch failed: reduceToBase_Transposed");
}
